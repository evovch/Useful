
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++) {
		y[i] = x[i] + y[i];
	}
}

int main(void)
{
	int N = 1<<18;
	float *x, *y;

	hipError_t rez1;
	hipError_t rez2;

	// Allocate Unified Memory – accessible from CPU or GPU
	rez1 = hipMallocManaged((void**)&x, N*sizeof(float));
	rez2 = hipMallocManaged((void**)&y, N*sizeof(float));

	std::cout << "rez1=" << rez1 << std::endl;
	std::cout << "rez2=" << rez2 << std::endl;

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// Run kernel on 1M elements on the GPU
	add<<<1, 1>>>(N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
	maxError = fmax(maxError, fabs(y[i]-3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}