
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

__device__
unsigned int Nmax = 10000;

template <typename TYPE>
__device__
TYPE myAbs(TYPE x)
{
	if (x >= static_cast<TYPE>(0.)) {
		return x;
	} else {
		return -x;
	}
}

template <typename TYPE>
__device__
TYPE func(TYPE x)
{
	return x*x*x + static_cast<TYPE>(2.3);
}

template <typename TYPE>
__device__
TYPE func_deriv(TYPE x)
{
	return static_cast<TYPE>(3.)*x*x;
}

template <typename TYPE>
__global__
void solve(TYPE epsilon, TYPE* x0, TYPE* xi)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	xi[j] = x0[j];
	for (unsigned int iIter=1; iIter<Nmax; iIter++) {
		TYPE diff = func(xi[j]) / func_deriv(xi[j]);
		//std::cout << "iteration " << iIter << "\t"
		//          << "diff=" << diff << std::endl;
		if (myAbs(diff) < epsilon) {
			return; // or 'break' to be more precise
		}
		xi[j] -= diff;
	}
}

template <typename TYPE>
void run(void)
{
	TYPE epsilon = static_cast<TYPE>(0.000001);
	//TYPE x0 = static_cast<TYPE>(200.)*uniRnd - static_cast<TYPE>(100.);

	TYPE* host_initVec;
	TYPE* host_solutionVec;
	TYPE* dev_initVec;
	TYPE* dev_solutionVec;

	unsigned int Nthreads = 128;

	host_initVec = (TYPE*)malloc(Nthreads*sizeof(TYPE));
	host_solutionVec = (TYPE*)malloc(Nthreads*sizeof(TYPE));
	hipError_t err1 = hipMalloc(&dev_initVec, Nthreads*sizeof(TYPE));
	hipError_t err2 = hipMalloc(&dev_solutionVec, Nthreads*sizeof(TYPE));

	if (err1 || err2) {
		std::cerr << "Failed to allocate memory." << std::endl;
		exit(EXIT_FAILURE);
	}

	for (unsigned int i=0; i<Nthreads; i++) {
		TYPE uniRnd = static_cast<TYPE>(rand()) / static_cast<TYPE>(RAND_MAX);
		host_initVec[i] = static_cast<TYPE>(200.)*uniRnd - static_cast<TYPE>(100.);
	}

	hipMemcpy(dev_initVec, host_initVec, Nthreads*sizeof(TYPE), hipMemcpyHostToDevice);

	solve<<<4,32>>>(epsilon, dev_initVec, dev_solutionVec);

	//cudaDeviceSynchronize();

	hipMemcpy(host_solutionVec, dev_solutionVec, Nthreads*sizeof(TYPE), hipMemcpyDeviceToHost);

	for (unsigned int i=0; i<Nthreads; i++) {
		std::cout << "x0[" << i << "]=" << host_initVec[i] << "\t"
		          << " x[" << i << "]=" << host_solutionVec[i]
		          << std::endl;
	}

	hipFree(dev_initVec);
	hipFree(dev_solutionVec);
	free(host_initVec);
	free(host_solutionVec);
}

int main(int argc, char** argv)
{
	srand(time(NULL));
	run<float>();
	return 0;
}
